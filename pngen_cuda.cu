#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <png.h>

#include "pngr.h"
#include "pngen.h"
#include "pngen_cuda.h"

/* util */
#define checkCudaErrors(val) check( (val), #val, __FILE__, __LINE__)

template<typename T>
void check(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    std::cerr << "CUDA error at: " << file << ":" << line << std::endl;
    std::cerr << hipGetErrorString(err) << " " << func << std::endl;
    exit(1);
  }
}


/* kernel for pixel generation */
__global__
void cudapixgen_sin(png_bytep pixel_bytes, int width, int height, int bpp) {
	const int2 thread_2D_pos = make_int2(blockIdx.x * blockDim.x + threadIdx.x,
		blockIdx.y * blockDim.y + threadIdx.y);

	int absArrayPos = thread_2D_pos.y * (width * bpp) + thread_2D_pos.x;
	if (thread_2D_pos.x > (width * bpp - 1) || thread_2D_pos.y > (height - 1))
		return;

	int x = thread_2D_pos.x,
		y = thread_2D_pos.y;

	/* t, s = normalized x, y */
	float t = 2.0 * M_PI * ((float) x / (float) (width * bpp)),
		s = 2.0 * M_PI * ((float) y / (float) height);

	/* f(t,s) = <r(t,s), g(t,s), b(t,s)>
	 * which component is the thread working on?
	 * Take horizontal position (resets with every new row)
	 * % 3 -> {0, 1, 2}
	 * 0: red
	 * 1: green
	 * 2: blue
	 */
	float pixelComp;
	switch (x % 3) {
		case 0:
			/* red */
			pixelComp = (127.0f * sin(t + tan(s - t)) + 128.0f);
			break;
		case 1:
			/* green */
			pixelComp = (127.0f * sin(s + tan(t - s)) + 128.0f);
			break;
		case 2:
			/* blue */
			pixelComp = (127.0f * sin(t*s) + 128.0f);
			break;
	}
	pixel_bytes[absArrayPos] = (png_byte) pixelComp;
}

void cudagen_png_sin(png_bytep *pixels, struct image_info *info) {
	/* h_pixels is a pointer to row pointers
	 * we still have to allocate memory for the individual pixels
	 */
	size_t numPixelBytes = info->row_size * info->height;
	png_bytep h_pixel_bytes = (png_bytep) malloc(numPixelBytes);
	if (h_pixel_bytes == NULL)
		gtfo("Unable to allocate sufficient memory");

	/* modify row pointer to point to these rows */
	int rown;
	for (rown = 0; rown < info->height; rown++)
		pixels[rown] = &(h_pixel_bytes[rown * info->row_size]);


	/* pointer to pixel array on device */
	png_bytep d_pixel_bytes;

	/* Allocate memory on GPU */
	checkCudaErrors(hipMalloc(&d_pixel_bytes, numPixelBytes));

	/* Block sizing */
	const dim3 block_size(48, 16, 1);

	/* Grid sizing */
	const dim3 grid_size((info->width/16) + 1, (info->height/16) + 1, 1);

	/* Launch kernel */
	cudapixgen_sin<<<grid_size, block_size>>>(d_pixel_bytes, info->width, info->height, info->bpp);

	/* wait for kernel to complete execution, then check for errors */
	hipDeviceSynchronize(); checkCudaErrors(hipGetLastError());

	/* Copy results from device -> host */
	checkCudaErrors(hipMemcpy(h_pixel_bytes, d_pixel_bytes, numPixelBytes, hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_pixel_bytes));
}